#include "hip/hip_runtime.h"
#include <assert.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

extern "C" {
#include "kmeans.h"
}
#include "kmeans_config.h"

/* Helper Functions ***********************************************************/

#define cudaAssert(code, file, line) do { \
  if (code != hipSuccess) { \
    fprintf(stderr, "A CUDA error occurred: %s (%s:%d)\n", \
            hipGetErrorString(code), file, line); \
    exit(code); \
  } \
} while(0)

#define cudaCheck(code) do { cudaAssert(code, __FILE__, __LINE__); } while(0)

/* CUDA kernels ***************************************************************/

// reassign points to closest centroids (#threads must be a power of two)
__global__
static void reassign(struct pixel *pixels, size_t n_pixels,
                     struct pixel *centroids, size_t n_centroids,
                     size_t *labels, struct pixel *sums, size_t *counts,
                     int *empty, int *done)
{
    // index alias
    size_t tid = threadIdx.x;
    size_t bid = blockIdx.x;

    // set up shared and global memory
    extern __shared__ char shared[];

    struct pixel *shared_pixels = (struct pixel *) shared;

    size_t shared_counts_offs = blockDim.x * sizeof(struct pixel);
    shared_counts_offs += sizeof(size_t) - sizeof(struct pixel) % sizeof(size_t);

    size_t *shared_counts = (size_t *) &shared[shared_counts_offs];

    if (tid < n_centroids) {
      shared_pixels[tid] = centroids[tid];

      struct pixel tmp = { 0.0, 0.0, 0.0 };
      sums[n_centroids * bid + tid] = tmp;
      counts[n_centroids * bid + tid] = 0u;
    }

    __syncthreads();

    // begin reassignment
    size_t index = bid * blockDim.x + tid;
    size_t closest_centroid = 0u;

    if (index >= n_pixels) {
       closest_centroid = n_centroids + 1u;
    } else {
        // find closest centroid
        double min_dist = DBL_MAX;

        struct pixel *p = &pixels[index];
        for (size_t j = 0u; j < n_centroids; ++j) {
            struct pixel *c = &shared_pixels[j];

            double dr = p->r - c->r;
            double dg = p->g - c->g;
            double db = p->b - c->b;

            double dist = sqrt(dr * dr + dg * dg + db * db);

            if (dist < min_dist) {
                closest_centroid = j;
                min_dist = dist;
            }
        }

        // if pixel has changed cluster...
        if (closest_centroid != labels[index]) {
            labels[index] = closest_centroid;

            *done = 0;
        }
    }

    // perform cluster wise tree-reduction to obtain cluster sums / counts
    for (size_t j = 0u; j < n_centroids; ++j) {
        if (j == closest_centroid) {
            shared_pixels[tid] = pixels[index];
            shared_counts[tid] = 1u;
            empty[j] = 0u;
        } else {
            struct pixel tmp = { 0.0, 0.0, 0.0 };
            shared_pixels[tid] = tmp;
            shared_counts[tid] = 0u;
        }

        __syncthreads();

        for (size_t dist = blockDim.x >> 1; dist > 0u; dist >>= 1) {
            if (tid < dist) {
                struct pixel *shared_sum1 = &shared_pixels[tid];
                struct pixel *shared_sum2 = &shared_pixels[tid + dist];

                shared_sum1->r += shared_sum2->r;
                shared_sum1->g += shared_sum2->g;
                shared_sum1->b += shared_sum2->b;

                shared_counts[tid] += shared_counts[tid + dist];
            }

            __syncthreads();
        }

        if (tid == 0) {
            struct pixel *sum = &sums[n_centroids * bid + j];
            struct pixel *shared_sum = &shared_pixels[0];

            sum->r += shared_sum->r;
            sum->g += shared_sum->g;
            sum->b += shared_sum->b;

            counts[n_centroids * bid + j] += shared_counts[0];
        }
    }
}

// reduce per-block cluster sums and counts
__device__
static void _reduce(size_t n_blocks, size_t n_centroids,
                    struct pixel *sums, size_t *counts)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    // reduce per-block clusters sums / counts
    for (size_t dist = (n_blocks * n_centroids) >> 1;
         dist >= n_centroids; dist >>= 1) {

        for (size_t i = index; i < dist; i += stride) {
            struct pixel *sum1 = &sums[i];
            struct pixel *sum2 = &sums[i + dist];

            sum1->r += sum2->r;
            sum1->g += sum2->g;
            sum1->b += sum2->b;

            counts[index] += counts[i + dist];
       }

       __syncthreads();
    }
}

__global__
static void reduce(size_t n_blocks, size_t n_centroids,
                   struct pixel *sums, size_t *counts)
{
    _reduce(n_blocks, n_centroids, sums, counts);
}

// re-calculate centroids
__global__
static void average(size_t n_blocks, struct pixel *centroids, size_t n_centroids,
                    struct pixel *sums, size_t *counts, int reduce)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    if (reduce)
        _reduce(n_blocks, n_centroids, sums, counts);

    // compute new centroids
    for (size_t i = index; i < n_centroids; i += stride) {
        struct pixel *c = &centroids[i];
        struct pixel *sum = &sums[i];
        size_t count = counts[i];

        c->r = sum->r / count;
        c->g = sum->g / count;
        c->b = sum->b / count;
    }
}

/* Main Function **************************************************************/

extern "C" void kmeans_cuda(struct pixel *pixels, size_t n_pixels,
                            struct pixel *centroids, size_t n_centroids,
                            size_t *labels)
{
    // number of blocks to be used on device
    size_t n_blocks_reassign =
        (n_pixels + KMEANS_CUDA_BLOCKSIZE - 1u) / KMEANS_CUDA_BLOCKSIZE;

    // round upwards to power of two
    size_t n_blocks_reassign_log2 = 0u;
    while(n_blocks_reassign >>= 1)
        ++n_blocks_reassign_log2;

    n_blocks_reassign = 1 << (n_blocks_reassign_log2 + 1u);

    size_t n_block_reduce =
        (((n_centroids * n_blocks_reassign) >> 1) + KMEANS_CUDA_BLOCKSIZE - 1u) /
        KMEANS_CUDA_BLOCKSIZE;

    // reassignment step shared memory size
    size_t shm_slots;
    if (n_centroids > KMEANS_CUDA_BLOCKSIZE)
        shm_slots = n_centroids;
    else
        shm_slots = KMEANS_CUDA_BLOCKSIZE;

    size_t shm_reassign = shm_slots * (sizeof(struct pixel) + sizeof(size_t));
    shm_reassign += sizeof(size_t) - sizeof(struct pixel) % sizeof(size_t);

    // initialize centroids with random pixels
    srand(time(NULL));

    for (size_t i = 0u; i < n_centroids; ++i)
        centroids[i] = pixels[rand() % n_pixels];

    // initialize device memory
    size_t pixels_sz = n_pixels * sizeof(struct pixel);
    size_t centroids_sz = n_centroids * sizeof(struct pixel);
    size_t labels_sz = n_pixels * sizeof(size_t);

    struct pixel *pixels_dev;
    struct pixel *centroids_dev;
    size_t *labels_dev;

    cudaCheck(hipMalloc(&pixels_dev, pixels_sz));
    cudaCheck(hipMalloc(&centroids_dev, centroids_sz));
    cudaCheck(hipMalloc(&labels_dev, labels_sz));

    cudaCheck(hipMemcpy(pixels_dev, pixels, pixels_sz,
                         hipMemcpyHostToDevice));

    cudaCheck(hipMemcpy(centroids_dev, centroids, centroids_sz,
                         hipMemcpyHostToDevice));

    cudaCheck(hipMemcpy(labels_dev, labels, labels_sz,
                         hipMemcpyHostToDevice));

    // allocate and initialize auxiliary memory
    size_t sums_sz = n_centroids * sizeof(struct pixel);
    size_t counts_sz = n_centroids * sizeof(size_t);
    size_t empty_sz = n_centroids * sizeof(int);
    size_t sums_dev_sz = n_blocks_reassign * n_centroids * sizeof(struct pixel);
    size_t counts_dev_sz = n_blocks_reassign * n_centroids * sizeof(size_t);

    struct pixel *sums, *sums_dev;
    size_t *counts, *counts_dev;
    int *empty, *empty_dev;
    int done, *done_dev;

    sums = (struct pixel *) malloc(sums_sz);
    counts = (size_t *) malloc(counts_sz);
    empty = (int *) malloc(empty_sz);

    cudaCheck(hipMalloc(&sums_dev, sums_dev_sz));
    cudaCheck(hipMalloc(&counts_dev, counts_dev_sz));
    cudaCheck(hipMalloc(&empty_dev, empty_sz));
    cudaCheck(hipMalloc(&done_dev, sizeof(int)));

    for (size_t i = 0u; i < n_centroids; ++i) {
        struct pixel tmp = { 0.0, 0.0, 0.0 };
        sums[i] = tmp;
        counts[i] = 0u;
    }

    // repeat for KMEANS_MAX_ITER or until solution is stationary
    for (int iter = 0; iter < KMEANS_MAX_ITER; ++iter) {
        for (size_t i = 0u; i < n_centroids; ++i)
            empty[i] = 1;

        done = 1;

        cudaCheck(hipMemcpy(empty_dev, empty, empty_sz,
                             hipMemcpyHostToDevice));

        cudaCheck(hipMemcpy(done_dev, &done, sizeof(int),
                             hipMemcpyHostToDevice));

        // reassign points to closest centroids
        reassign<<<n_blocks_reassign, KMEANS_CUDA_BLOCKSIZE, shm_reassign>>>(
            pixels_dev, n_pixels, centroids_dev, n_centroids, labels_dev,
            sums_dev, counts_dev, empty_dev, done_dev
        );

        cudaCheck(hipPeekAtLastError());

        cudaCheck(hipMemcpy(empty, empty_dev, empty_sz,
                             hipMemcpyDeviceToHost));

        cudaCheck(hipMemcpy(&done, done_dev, sizeof(int),
                             hipMemcpyDeviceToHost));

        // check whether empty clusters need to be repaired
        int repair = 0;
        for (size_t i = 0u; i < n_centroids; ++i) {
            if (!empty[i])
                continue;

            // reduce in separate kernel
            reduce<<<n_block_reduce, KMEANS_CUDA_BLOCKSIZE>>>(
                n_blocks_reassign, n_centroids, sums_dev, counts_dev
            );

            cudaCheck(hipMemcpy(sums, sums_dev, sums_sz,
                                 hipMemcpyDeviceToHost));

            cudaCheck(hipMemcpy(counts, counts_dev, counts_sz,
                                 hipMemcpyDeviceToHost));

            done = 0;
            repair = 1;
            break;
        }

        // repair empty clusters (on host)
        if (repair) {
            for (size_t i = 0u; i < n_centroids; ++i) {
                if (!empty[i])
                    continue;

                done = 0;

                // determine largest cluster
                size_t largest_cluster = 0u;
                size_t largest_cluster_count = 0u;
                for (size_t j = 0u; j < n_centroids; ++j) {
                    if (j == i)
                        continue;

                    if (counts[j] > largest_cluster_count) {
                        largest_cluster = j;
                        largest_cluster_count = counts[j];
                    }
                }

                // determine pixel in this cluster furthest from its centroid
                struct pixel *largest_cluster_centroid =
                    &centroids[largest_cluster];

                size_t furthest_pixel = 0u;
                double max_dist = 0.0;
                for (size_t j = 0u; j < n_pixels; ++j) {
                    if (labels[j] != largest_cluster)
                        continue;

                    struct pixel *p = &pixels[j];

                    double dr = p->r - largest_cluster_centroid->r;
                    double dg = p->g - largest_cluster_centroid->g;
                    double db = p->b - largest_cluster_centroid->b;

                    double dist = sqrt(dr * dr + dg * dg + db * db);

                    if (dist > max_dist) {
                        furthest_pixel = j;
                        max_dist = dist;
                    }
                }

                // move that pixel to the empty cluster
                struct pixel replacement_pixel = pixels[furthest_pixel];
                centroids[i] = replacement_pixel;
                labels[furthest_pixel] = i;

                // correct cluster sums
                sums[i] = replacement_pixel;

                struct pixel *sum = &sums[largest_cluster];
                sum->r -= replacement_pixel.r;
                sum->g -= replacement_pixel.g;
                sum->b -= replacement_pixel.b;

                // correct cluster sizes
                counts[i] = 1u;
                counts[largest_cluster]--;
            }

            cudaCheck(hipMemcpy(sums_dev, sums, sums_sz,
                                 hipMemcpyHostToDevice));

            cudaCheck(hipMemcpy(counts_dev, counts, counts_sz,
                                 hipMemcpyHostToDevice));
        }

        // re-calculate centroids
        average<<<n_block_reduce, KMEANS_CUDA_BLOCKSIZE>>>(
            n_blocks_reassign, centroids_dev, n_centroids,
            sums_dev, counts_dev, !repair
        );

        cudaCheck(hipPeekAtLastError());
        cudaCheck(hipDeviceSynchronize());

        // break if no pixel has changed cluster
        if (done)
            break;
    }

    // copy device memory back to host
    cudaCheck(hipMemcpy(pixels, pixels_dev, pixels_sz,
                         hipMemcpyDeviceToHost));

    cudaCheck(hipMemcpy(centroids, centroids_dev, centroids_sz,
                         hipMemcpyDeviceToHost));

    cudaCheck(hipMemcpy(labels, labels_dev, labels_sz,
                         hipMemcpyDeviceToHost));

    // free host and device memory
    free(sums);
    free(counts);

    cudaCheck(hipFree(pixels_dev));
    cudaCheck(hipFree(centroids_dev));
    cudaCheck(hipFree(labels_dev));
    cudaCheck(hipFree(sums_dev));
    cudaCheck(hipFree(counts_dev));
}
