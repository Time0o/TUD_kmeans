#include "hip/hip_runtime.h"
#include <cassert>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

extern "C" {
#include "kmeans.h"
}
#include "kmeans_config.h"

/* Helper Functions ***********************************************************/

#define cudaAssert(code, file, line) do {                  \
  if (code != hipSuccess) {                               \
    fprintf(stderr, "A CUDA error occurred: %s (%s:%d)\n", \
            hipGetErrorString(code), file, line);         \
    exit(code);                                            \
  }                                                        \
} while(0)

#define cudaCheck(code) do { cudaAssert(code, __FILE__, __LINE__); } while(0)

/* CUDA kernels ***************************************************************/

// reassign points to closest centroids
__global__
static void reassign(struct pixel *pixels, size_t n_pixels,
                     struct pixel *centroids, size_t n_centroids,
                     size_t *labels, struct pixel *sums, size_t *counts,
                     int *done)
{
    // index alias
    size_t tid = threadIdx.x;
    size_t bid = blockIdx.x;

    // set up shared and global memory
    extern __shared__ char shared[];

    struct pixel *shared_pixels = (struct pixel *) shared;

    size_t shared_counts_offs = blockDim.x * sizeof(struct pixel);
    shared_counts_offs += sizeof(size_t) - sizeof(struct pixel) % sizeof(size_t);

    size_t *shared_counts = (size_t *) &shared[shared_counts_offs];

    if (tid < n_centroids) {
      shared_pixels[tid] = centroids[tid];

      struct pixel tmp = { 0.0, 0.0, 0.0 };
      sums[n_centroids * bid + tid] = tmp;

      counts[n_centroids * bid + tid] = 0u;
    }

    __syncthreads();

    // obtain pixel index and stride (less threads than total pixels available)
    size_t index = bid * blockDim.x + tid;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < n_pixels; i += stride) {
        struct pixel *p = &pixels[i];

        // find centroid closest to pixel
        size_t closest_centroid = 0u;
        double min_dist = DBL_MAX;

        for (size_t j = 0u; j < n_centroids; ++j) {
            struct pixel *c = &shared_pixels[j];

            double dr = p->r - c->r;
            double dg = p->g - c->g;
            double db = p->b - c->b;

            double dist = sqrt(dr * dr + dg * dg + db * db);

            if (dist < min_dist) {
                closest_centroid = j;
                min_dist = dist;
            }
        }

        // if pixel has changed cluster...
        if (closest_centroid != labels[i]) {
            labels[i] = closest_centroid;

            *done = 0;
        }

        // perform cluster wise tree-reduction to obtain cluster sums / counts
        for (size_t j = 0u; j < n_centroids; ++j) {
            if (j == closest_centroid) {
                shared_pixels[tid] = *p;
                shared_counts[tid] = 1u;
            } else {
                struct pixel tmp = { 0.0, 0.0, 0.0 };
                shared_pixels[tid] = tmp;
                shared_counts[tid] = 0u;
            }

            __syncthreads();

            for (size_t dist = blockDim.x >> 1; dist > 0; dist >>= 1) {
                if (tid < dist) {
                    struct pixel *shared_sum1 = &shared_pixels[tid];
                    struct pixel *shared_sum2 = &shared_pixels[tid + dist];

                    shared_sum1->r += shared_sum2->r;
                    shared_sum1->g += shared_sum2->g;
                    shared_sum1->b += shared_sum2->b;

                    shared_counts[tid] += shared_counts[tid + dist];
                }

                __syncthreads();
            }

            if (tid == 0) {
                struct pixel *sum = &sums[n_centroids * bid + j];
                struct pixel *shared_sum = &shared_pixels[0];

                sum->r += shared_sum->r;
                sum->g += shared_sum->g;
                sum->b += shared_sum->b;

                counts[n_centroids * bid + j] += shared_counts[0];
            }
        }
    }
}

// reduce per-block cluster sums and counts
__global__
static void average(size_t n_blocks, struct pixel *centroids, size_t n_centroids,
                    struct pixel *sums, size_t *counts)
{
    size_t tid = threadIdx.x;
    size_t bid = blockIdx.x;

    size_t index = bid * blockDim.x + tid;
    size_t stride = blockDim.x * gridDim.x;

    // reduce per-block clusters sums / counts
    for (size_t dist = (n_blocks * n_centroids) >> 1; dist >= n_centroids; dist >>= 1) {
        for (size_t i = index; i < dist; i += stride) {
            struct pixel *sum1 = &sums[i];
            struct pixel *sum2 = &sums[i + dist];

            sum1->r += sum2->r;
            sum1->g += sum2->g;
            sum1->b += sum2->b;

            counts[i] += counts[i + dist];
        }
        __syncthreads();
    }

    // compute new centroids
    for (size_t j = index; j < n_centroids; j += stride) {
        struct pixel *c = &centroids[j];
        struct pixel *sum = &sums[j];
        size_t count = counts[j];

        c->r = sum->r / count;
        c->g = sum->g / count;
        c->b = sum->b / count;
    }
}

/* Main Function **************************************************************/

extern "C" void kmeans_cuda(struct pixel *pixels, size_t n_pixels,
                            struct pixel *centroids, size_t n_centroids,
                            size_t *labels)
{
    // number of blocks to be used on device
    size_t n_blocks_reassign =
        (n_pixels + KMEANS_CUDA_BLOCKSIZE - 1) / KMEANS_CUDA_BLOCKSIZE;

    size_t n_blocks_average =
        (((n_centroids * n_blocks_reassign) >> 1) + KMEANS_CUDA_BLOCKSIZE - 1) /
        KMEANS_CUDA_BLOCKSIZE;

    // reassignment step shared memory size
    size_t shm_slots =
        n_centroids > KMEANS_CUDA_BLOCKSIZE ? n_centroids : KMEANS_CUDA_BLOCKSIZE;

    size_t shm_reassign = shm_slots * (sizeof(struct pixel) + sizeof(size_t));
    shm_reassign += sizeof(size_t) - sizeof(struct pixel) % sizeof(size_t);

    // initialize centroids with random pixels
    srand(time(NULL));

    for (size_t i = 0u; i < n_centroids; ++i)
        centroids[i] = pixels[rand() % n_pixels];

    // initialize device memory
    struct pixel *pixels_dev;
    struct pixel *centroids_dev;
    size_t *labels_dev;

    cudaCheck(hipMalloc(&pixels_dev, n_pixels * sizeof(struct pixel)));
    cudaCheck(hipMalloc(&centroids_dev, n_centroids * sizeof(struct pixel)));
    cudaCheck(hipMalloc(&labels_dev, n_pixels * sizeof(size_t)));

    cudaCheck(hipMemcpy(pixels_dev, pixels, n_pixels * sizeof(struct pixel),
                         hipMemcpyHostToDevice));

    cudaCheck(hipMemcpy(centroids_dev, centroids, n_centroids * sizeof(struct pixel),
                         hipMemcpyHostToDevice));

    cudaCheck(hipMemcpy(labels_dev, labels, n_pixels * sizeof(size_t),
                         hipMemcpyHostToDevice));

    // allocate and initialize auxiliary memory
    struct pixel *sums, *sums_dev;
    size_t *counts, *counts_dev;
    int done, *done_dev;

    sums = (struct pixel *) malloc(n_centroids * sizeof(struct pixel));
    counts = (size_t *) malloc(n_centroids *  sizeof(size_t));

    cudaCheck(hipMalloc(&sums_dev, n_blocks_reassign * n_centroids * sizeof(struct pixel)));
    cudaCheck(hipMalloc(&counts_dev, n_blocks_reassign * n_centroids *  sizeof(size_t)));
    cudaCheck(hipMalloc(&done_dev, sizeof(int)));

    for (size_t i = 0u; i < n_centroids; ++i) {
        struct pixel tmp = { 0.0, 0.0, 0.0 };
        sums[i] = tmp;

        counts[i] = 0u;
    }

    // repeat for KMEANS_MAX_ITER or until solution is stationary
    for (int iter = 0; iter < KMEANS_MAX_ITER; ++iter) {
        done = 1;

        cudaCheck(hipMemcpy(done_dev, &done, sizeof(int),
                             hipMemcpyHostToDevice));

        // reassign points to closest centroids
        reassign<<<n_blocks_reassign, KMEANS_CUDA_BLOCKSIZE, shm_reassign>>>(
            pixels_dev, n_pixels, centroids_dev, n_centroids, labels_dev,
            sums_dev, counts_dev, done_dev
        );

        cudaCheck(hipPeekAtLastError());
        cudaCheck(hipDeviceSynchronize());

        average<<<n_blocks_average, KMEANS_CUDA_BLOCKSIZE>>>(
            n_blocks_reassign, centroids_dev, n_centroids, sums_dev, counts_dev
        );

        cudaCheck(hipPeekAtLastError());
        cudaCheck(hipMemcpy(&done, done_dev, sizeof(int),
                             hipMemcpyDeviceToHost));

        // break if no pixel has changed cluster
        if (done)
            break;
    }

    // copy device memory back to host
    cudaCheck(hipMemcpy(pixels, pixels_dev, n_pixels * sizeof(struct pixel),
                         hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(centroids, centroids_dev, n_centroids * sizeof(struct pixel),
                         hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(labels, labels_dev, n_pixels * sizeof(size_t),
                         hipMemcpyDeviceToHost));

    // free host and device memory
    free(sums);
    free(counts);

    cudaCheck(hipFree(pixels_dev));
    cudaCheck(hipFree(centroids_dev));
    cudaCheck(hipFree(labels_dev));
    cudaCheck(hipFree(sums_dev));
    cudaCheck(hipFree(counts_dev));
}
