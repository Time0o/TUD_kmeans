#include "hip/hip_runtime.h"
#include <float.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

extern "C" {
#include "kmeans.h"
}
#include "kmeans_config.h"

// reassign points to closest centroids
__global__
static void reassign_points(struct pixel *pixels, size_t n_pixels,
                            struct pixel *centroids, size_t n_centroids,
                            size_t *labels, struct pixel *sums, size_t *counts,
                            int *done)
{
  int index = blockIdx.x * blockDim.x * threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n_pixels; i += stride) {
      struct pixel pixel = pixels[i];

      // find centroid closest to pixel
      size_t closest_centroid = 0u;
      double min_dist = DBL_MAX;

      for (size_t j = 0; j < n_centroids; ++j) {
          struct pixel centroid = centroids[j];

          double dr = pixel.r - centroid.r;
          double dg = pixel.g - centroid.g;
          double db = pixel.b - centroid.b;

          double dist = sqrt(dr * dr + dg * dg + db * db);

          if (dist < min_dist) {
              closest_centroid = j;
              min_dist = dist;
          }
      }

      // if pixel has changed cluster...
      if (closest_centroid != labels[i]) {
          labels[i] = closest_centroid;

          *done = 0;
      }

      // update cluster sum
      struct pixel *sum = &sums[closest_centroid];
      sum->r += pixel.r;
      sum->g += pixel.g;
      sum->b += pixel.b;

      // update cluster size
      counts[closest_centroid]++;
  }
}

// compute euclidean distance between two pixel values
static inline double pixel_dist(struct pixel p1, struct pixel p2)
{
    double dr = p1.r - p2.r;
    double dg = p1.g - p2.g;
    double db = p1.b - p2.b;

    return sqrt(dr * dr + dg * dg + db * db);
}

extern "C" void kmeans_cuda(struct pixel *pixels, size_t n_pixels,
                            struct pixel *centroids, size_t n_centroids,
                            size_t *labels)
{
    // number of blocks to be used on device
    int blocks = (n_pixels + KMEANS_CUDA_BLOCKSIZE - 1) / KMEANS_CUDA_BLOCKSIZE;

    // initialize device memory
    struct pixel *pixels_dev;
    struct pixel *centroids_dev;
    size_t *labels_dev;

    hipMalloc(&pixels_dev, n_pixels * sizeof(struct pixel));
    hipMalloc(&centroids_dev, n_centroids * sizeof(struct pixel));
    hipMalloc(&labels_dev, n_pixels * sizeof(size_t));

    hipMemcpy(pixels_dev, pixels, n_pixels * sizeof(struct pixel),
               hipMemcpyHostToDevice);
    hipMemcpy(centroids_dev, centroids, n_centroids * sizeof(struct pixel),
               hipMemcpyHostToDevice);
    hipMemcpy(labels_dev, labels, n_pixels * sizeof(size_t),
               hipMemcpyHostToDevice);

    // seed rand
    srand(time(NULL));

    // allocate auxiliary memory shared between host and device
    struct pixel *sums, *sums_dev;
    size_t *counts, *counts_dev;

    sums = malloc(n_centroids * sizeof(struct pixel));
    counts = malloc(n_centroids *  sizeof(size_t));

    hipMalloc(&sums_dev, n_centroids * sizeof(struct pixel));
    hipMalloc(&counts_dev, n_centroids *  sizeof(size_t));

    // randomly initialize centroids
    for (size_t i = 0u; i < n_centroids; ++i) {
        centroids[i] = pixels[rand() % n_pixels];

        struct pixel tmp = { 0.0, 0.0, 0.0 };
        sums[i] = tmp;

        counts[i] = 0u;
    }

    // repeat for KMEANS_MAX_ITER or until solution is stationary
    int iter;
    for (iter = 0; iter < KMEANS_MAX_ITER; ++iter) {
        int *done;
        hipMallocManaged(&done, sizeof(int));
        *done = 1;

        // reassign points to closest centroids
        reassign_points<<<blocks, KMEANS_CUDA_BLOCKSIZE>>>(
            pixels_dev, n_pixels, centroids_dev, n_centroids, labels_dev,
            sums, counts, done);

        // repair empty clusters
        for (size_t i = 0u; i < n_centroids; ++i) {
            if (counts[i])
                continue;

            // determine largest cluster
            size_t largest_cluster = 0u;
            size_t largest_cluster_count = 0u;
            for (size_t j = 0u; j < n_centroids; ++j) {
                if (j == i)
                    continue;

                if (counts[j] > largest_cluster_count) {
                    largest_cluster = j;
                    largest_cluster_count = counts[j];
                }
            }

            // determine pixel in this cluster furthest from its centroid
            struct pixel largest_cluster_centroid = centroids[largest_cluster];

            size_t furthest_pixel = 0u;
            double max_dist = 0.0;
            for (size_t j = 0u; j < n_pixels; ++j) {
                if (labels[j] != largest_cluster)
                    continue;

                double dist = pixel_dist(pixels[j], largest_cluster_centroid);

                if (dist > max_dist) {
                    furthest_pixel = j;
                    max_dist = dist;
                }
            }

            // move that pixel to the empty cluster
            struct pixel replacement_pixel = pixels[furthest_pixel];
            centroids[i] = replacement_pixel;
            labels[furthest_pixel] = i;

            // correct cluster sums
            sums[i] = replacement_pixel;

            struct pixel *sum = &sums[largest_cluster];
            sum->r -= replacement_pixel.r;
            sum->g -= replacement_pixel.g;
            sum->b -= replacement_pixel.b;

            // correct cluster sizes
            counts[i] = 1u;
            counts[largest_cluster]--;
        }

        // average accumulated cluster sums
        for (int j = 0; j < n_centroids; ++j) {
            struct pixel *centroid = &centroids[j];
            struct pixel *sum = &sums[j];
            size_t count = counts[j];

            centroid->r = sum->r / count;
            centroid->g = sum->g / count;
            centroid->b = sum->b / count;

            sum->r = 0.0;
            sum->g = 0.0;
            sum->b = 0.0;

            counts[j] = 0u;
        }

        // break if no pixel has changed cluster
        if (*done)
            break;
    }

    // copy device memory back to host
    hipMemcpy(pixels, pixels_dev, n_pixels * sizeof(struct pixel),
               hipMemcpyDeviceToHost);
    hipMemcpy(centroids, centroids_dev, n_centroids * sizeof(struct pixel),
               hipMemcpyDeviceToHost);
    hipMemcpy(labels, labels_dev, n_pixels * sizeof(size_t),
               hipMemcpyDeviceToHost);

    // free host and device memory
    hipFree(pixels_dev);
    hipFree(centroids_dev);
    hipFree(labels_dev);
    hipFree(sums);
    hipFree(counts);
}
